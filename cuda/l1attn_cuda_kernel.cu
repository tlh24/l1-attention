#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/native/cuda/KernelUtils.cuh>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

template <typename scalar_t>
__device__  __forceinline__ scalar_t sign(scalar_t x)
{ 
	scalar_t t = x < 0 ? -1 : 0;
	return x > 0 ? 1 : t;
}

template <typename scalar_t>
__device__  __forceinline__ void fastAtomicAdd2(
	torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> out, 
	int i0, int i1, int i2, int i3, scalar_t v)
{
	// convenience wrapper function around
	// fastAtomicAdd for 4-D tensors. 
	int index = i0*out.stride(0) + i1*out.stride(1) + i2*out.stride(2) + i3*out.stride(3);
	at::native::fastAtomicAdd(out.data(), index, 1, v, true); 
}

// Optimisation: using shared memory for q and k to reduce global memory accesses
template <typename scalar_t>
__global__ void l1attn_cuda_forward_kernelX(
		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		q,
		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		k,
		torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		attn,
		const scalar_t scale, 
		const int bs, const int n_ctx, const int n_heads, const int width)
{
	__shared__ scalar_t acc[32];
	__shared__ scalar_t q_shared[32];
	__shared__ scalar_t k_shared[32];
	
	int tix = threadIdx.x; // [0 .. 31]. 
	// tix operates within across the width dimension (reduction dim) 
	int h = blockIdx.x % n_heads; 
	int t = blockIdx.x / n_heads; 
	int s = blockIdx.y; 
	int b = blockIdx.z; 
	
	int width32 = (width + 31) / 32; 
	scalar_t f = 0.0; 
	for(int w = 0; w < width32; w++) { 
		int o = w*32+tix; 
		if(o < width) {
			q_shared[tix] = q[b][t][h][o];
			k_shared[tix] = k[b][s][h][o];
			f += abs(q_shared[tix] - k_shared[tix]); 
		}
	}
	acc[tix] = f * scale; 
	if(tix < 16) { 
		acc[tix] += acc[tix + 16];
		__syncthreads(); // why is this needed ??? 
		acc[tix] += acc[tix + 8 ];
		__syncthreads(); // threads in a warp should be synchronous.
		acc[tix] += acc[tix + 4 ];
		__syncthreads(); // experiment: it's totally needed! 
		acc[tix] += acc[tix + 2 ];
		__syncthreads();
		acc[tix] += acc[tix + 1 ];
		__syncthreads();
		if(tix == 0){
			attn[b][s][t][h] = acc[tix]; 
		}
	}
}

#define	BLKSIZ 16
// Optimisation: Using shared memory for q and k. also perform coalesced memory accesses
template <typename scalar_t>
__global__ void l1attn_cuda_forward_kernel16(
		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		q,
		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		k,
		torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		attn,
		const scalar_t scale, 
		const int bs, const int n_ctx, const int n_heads, const int width)
{
	// q and k must be bhtw and bhsw respectively
	// despite the name of this function, it only operates on 
	// width 32 q and k tensors, in blocks of 16 x 16
	// Larger would require more per-warp memory or use of registers: 
	// 2 x 16 x 32 x 4 bytes = 4096 kB per block, so each SM can have 12 blocks. 
	
	int w = threadIdx.x; // t thread [0 .. 15]. 
	int u = threadIdx.y; // t for q, s for k,  [0 .. 15]. 
	int tb = blockIdx.x; // t block
	int sb = blockIdx.y; // s block
	int h = blockIdx.z % n_heads; // head
	int b = blockIdx.z / n_heads; // block
	
	// each block computes a BLKSIZ x BLKSIZ block of the attention matrix
	// a block is 256 threads
	// so, each thread loads one value from each q,k
	__shared__ scalar_t qc[BLKSIZ][32]; // q cache 
	__shared__ scalar_t kc[BLKSIZ][32]; // k cache
	
	//reshape to 8 warps, 32 threads - better mem throughput
	int tid = u*BLKSIZ + w; 
	int cw = tid % 32; // cache w
	int cu = tid / 32; // cache u
	int t = tb * BLKSIZ + cu; 
	int s = sb * BLKSIZ + cu; 
	
	// Optimization: Perform coalesced memory accesses
	for (int i = 0; i < 2; i++) {
		qc[cu + i*8][cw] = q[b][h][t + i*8][cw];
		kc[cu + i*8][cw] = k[b][h][s + i*8][cw];
	}
	
	__syncthreads();
	
	// simple approach: each thread computes one attention value
	// redefine t and s
	t = u; // so q is shared between threads in the same warp
	s = w; 
	scalar_t f = 0.0; 
	for(int o=0; o < 32; o++){
		f += abs(qc[t][o] - kc[s][o]); // ultimately want these to be registers
	}
	// back to global
	t = tb * BLKSIZ + u; 
	s = sb * BLKSIZ + w; 
	attn[b][s][t][h] = f * scale; // this is unaligned. ought to fix.
}

// template <typename scalar_t>
// __global__ void l1attn_cuda_backward_kernel_old(
// 		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
// 		d_attn,
// 		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
// 		q,
// 		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
// 		k,
// 		torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
// 		d_q,
// 		torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
// 		d_k,
// 		const scalar_t scale, 
// 		const int bs, const int n_ctx, const int n_heads, const int width ) 
// {
// 	// reduction (across s and t) has to be done within a thread warp: 
// 	// can't have different warps write the same memory. 
// 	// they will interfere / not give the correct answer!
// 	
// 	int indx = threadIdx.x + blockIdx.x * blockDim.x; // 1D
// 	
// 	if(indx < bs*n_ctx*n_ctx*n_heads){
// 		// again, output indexing b/c thread blocks can't overlap writes.
// 		// see note in forward kernel.
// 		int j = indx; 
// 		int h = j % n_heads; 
// 		j /= n_heads; 
// 		int s = j % n_ctx; 
// 		j /= n_ctx; 
// 		int t = j % n_ctx; 
// 		j /= n_ctx; 
// 		int b = j % bs; 
// 		
// 		scalar_t d_a = d_attn[b][s][t][h]; 
// 		for(int w = 0; w < width; w++){
// 			scalar_t ws = q[b][t][h][w] - k[b][s][h][w];
// 			ws = sign(ws) * scale; 
// 			// atomicAdd((scalar_t*)&(d_q[b][t][h][w]), ws * d_a);
// 			// atomicAdd((scalar_t*)&(d_k[b][s][h][w]), -1*ws * d_a);
// 			fastAtomicAdd2(d_q, b,t,h,w, ws * d_a);
// 			fastAtomicAdd2(d_k, b,s,h,w, -1*ws * d_a);
// 		}
// 	}
// } 

// Optimisation: Using shared memory for q and k. also perform coalesced memory accesses
template <typename scalar_t>
__global__ void l1attn_cuda_backward_kernel(
		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		d_attnq,
		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		d_attnk,
		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		q,
		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		k,
		torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		d_q,
		torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		d_k,
		const scalar_t scale, 
		const int bs, const int n_ctx, const int n_heads, const int width ) 
{
	__shared__ scalar_t acc_dq[32];
	__shared__ scalar_t acc_dk[32];
	__shared__ scalar_t q_shared[32];
	__shared__ scalar_t k_shared[32];
	
	int tix = threadIdx.x; // [0 .. 31].
	int h = blockIdx.x % n_heads; 
	int r = blockIdx.x / n_heads; // r is t for q, s for k.
	int w = blockIdx.y; 
	int b = blockIdx.z; 
		
	int ctx32 = (n_ctx + 31) / 32; 
	scalar_t dq = 0.0; 
	scalar_t dk = 0.0; 
	
	q_shared[tix] = q[b][w][h][r];
	scalar_t qq = q_shared[tix];
	for(int o = 0; o < ctx32; o++) { 
		int s = o*32+tix; 
		if(s < n_ctx){ 
			// all this would work better if n_ctx were a multiple of 32. 
			k_shared[tix] = k[b][w][h][s];
			scalar_t ws = qq - k_shared[tix];
			ws = sign(ws) * scale; 
			scalar_t d_a = d_attnq[b][r][h][s]; 
			dq += ws * d_a; 
		}
	}
	
	k_shared[tix] = k[b][w][h][r];
	scalar_t kk = k_shared[tix];
	for(int o = 0; o < ctx32; o++) { 
		int t = o*32+tix; 
		if(t < n_ctx){
			q_shared[tix] = q[b][w][h][t];
			scalar_t ws = q_shared[tix] - kk;
			ws = sign(ws) * scale; 
			scalar_t d_a = d_attnk[b][r][h][t]; 
			dk -= ws * d_a; 
		}
	}
	
	acc_dq[tix] = dq;
	acc_dk[tix] = dk;
	if(tix < 16) { 
		acc_dq[tix] += acc_dq[tix + 16];
		acc_dk[tix] += acc_dk[tix + 16];
		__syncthreads(); 
		acc_dq[tix] += acc_dq[tix + 8 ];
		acc_dk[tix] += acc_dk[tix + 8 ];
		__syncthreads(); 
		acc_dq[tix] += acc_dq[tix + 4 ];
		acc_dk[tix] += acc_dk[tix + 4 ];
		__syncthreads();
		acc_dq[tix] += acc_dq[tix + 2 ];
		acc_dk[tix] += acc_dk[tix + 2 ];
		__syncthreads();
		acc_dq[tix] += acc_dq[tix + 1 ];
		acc_dk[tix] += acc_dk[tix + 1 ];
		__syncthreads();
		if(tix == 0){
			d_q[b][r][h][w] = acc_dq[tix];
			d_k[b][r][h][w] = acc_dk[tix]; 
		}
	}
}

// Optimisation: using the shared memory for q, k, and d_attn, and perform coalesced memory accesses
template <typename scalar_t>
__global__ void l1attn_cuda_backward_kernel16(
		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		d_attn,
		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		q,
		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		k,
		torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		d_q,
		torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		d_k,
		const scalar_t scale, 
		const int bs, const int n_ctx, const int n_heads, const int width) 
{
	// q and k must be bhtw and bhsw respectively
	// d_attn must be bhts (usually bsth)
	// output is bhtzw / bhszw, where z is an extra reduction dim over 16x16 
	
	int v = threadIdx.x; // thread [0 .. 15]. 
	int r = threadIdx.y; // t for q, s for k,  [0 .. 15]. 
	int sb = blockIdx.x; // s block
	int tb = blockIdx.y; // t block
	int h = blockIdx.z % n_heads; // head
	int b = blockIdx.z / n_heads; // block
	
	// each block computes a BLKSIZ x 32 block of d_q, d_k
	// a block is 256 threads
	// so, each thread loads four values from each q,k
	// and one from d_attn
	__shared__ scalar_t dac[BLKSIZ][BLKSIZ]; // d_attn cache 
	__shared__ scalar_t qc[BLKSIZ][32]; // q cache 
	__shared__ scalar_t kc[BLKSIZ][32]; // k cache
	
	// Optimisation: Perform coalesced memory accesses
	int s = sb * BLKSIZ + v; 
	int t = tb * BLKSIZ + r; 
	dac[r][v] = d_attn[b][h][t][s]; 
	
	int tid = r*BLKSIZ + v; 
	int cw = tid % 32; // cache w
	int cr = tid / 32; // cache r
	s = sb * BLKSIZ + cr; 
	t = tb * BLKSIZ + cr; 
	
	for (int i = 0; i < 2; i++) {
		qc[cr + i*8][cw] = q[b][h][t + i*8][cw];
		kc[cr + i*8][cw] = k[b][h][s + i*8][cw];
	}
	__syncthreads();
	
	scalar_t dq, dk, qq, kk;
	for(int p = 0; p < 32; p += 16){
		int w = v + p; 
		dq = 0.0;
		t = r; 
		qq = qc[t][w]; 
		for(s = 0; s < BLKSIZ; s++){
			scalar_t ws = qq - kc[s][w];
			ws = sign(ws) * scale; 
			dq += ws * dac[t][s]; 
		}
		t = tb * BLKSIZ + r;
		//d_q[b][t][h][z][w] = dq; 
		fastAtomicAdd2( d_q, b,h,t,w, dq ); // ouch. o/w need too much mem.
		
		dk = 0.0; 
		s = r; 
		kk = kc[s][w]; 
		for(t = 0; t < BLKSIZ; t++){
			scalar_t ws = qc[t][w] - kk;
			ws = sign(ws) * scale; 
			dk -= ws * dac[t][s]; 
		}
		s = sb * BLKSIZ + r; 
		//d_k[b][s][h][z][w] = dk; 
		fastAtomicAdd2( d_k, b,h,s,w, dk ); 
	}
}

std::vector<torch::Tensor> l1attn_cuda_forward(
		torch::Tensor q,
		torch::Tensor k) {
  
	int bs = q.sizes()[0]; 
	int n_ctx = q.sizes()[1]; 
	int n_heads = q.sizes()[2]; 
	int width = q.sizes()[3];
	
	auto options = torch::TensorOptions()
		.dtype(q.dtype())
		.device(q.device())
		.requires_grad(q.requires_grad()); //better way to do this? 
	
	auto attn = torch::zeros({bs, n_ctx, n_ctx, n_heads}, options); 
	
	const dim3 numBlocks(n_heads*n_ctx, n_ctx, bs); // x, y, z
	const dim3 threadsPerBlock(32, 1, 1);
	
	double scale = -1.0 / sqrt(width); 
		
	AT_DISPATCH_FLOATING_TYPES(q.scalar_type(), "l1attn_cuda_forward_kernel", ([&] {
		l1attn_cuda_forward_kernelX<scalar_t><<<numBlocks, threadsPerBlock>>>(
			q.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			k.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			attn.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			scale, bs, n_ctx, n_heads, width);
	}));
	
	return {attn};
}

std::vector<torch::Tensor> l1attn_cuda_forward16(
		torch::Tensor q,
		torch::Tensor k) {
  
	int bs = q.sizes()[0]; 
	int n_heads = q.sizes()[1];
	int n_ctx = q.sizes()[2]; 
	int width = q.sizes()[3];
	
	auto options = torch::TensorOptions()
		.dtype(q.dtype())
		.device(q.device())
		.requires_grad(q.requires_grad()); //better way to do this? 
	
	auto attn = torch::zeros({bs, n_ctx, n_ctx, n_heads}, options); 
	
	const dim3 numBlocks(n_ctx/BLKSIZ, n_ctx/BLKSIZ, bs*n_heads); // x, y, z
	const dim3 threadsPerBlock(BLKSIZ, BLKSIZ, 1);
	
	double scale = -1.0 / sqrt(width); 
		
	AT_DISPATCH_FLOATING_TYPES(q.scalar_type(), "l1attn_cuda_forward_kernel16", ([&] {
		l1attn_cuda_forward_kernel16<scalar_t><<<numBlocks, threadsPerBlock>>>(
			q.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			k.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			attn.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			scale, bs, n_ctx, n_heads, width);
	}));
	
	// output is bhts; should be bsth to work with everything else.
	// attn = attn.transpose(1,3).contiguous(); 
	
	return {attn};
}

std::vector<torch::Tensor> l1attn_cuda_backward(
		torch::Tensor d_attnq,
		torch::Tensor d_attnk,
		torch::Tensor q,
		torch::Tensor k) 
{
	int bs = q.sizes()[0]; // permuted in python driver!!!
	int width = q.sizes()[1];
	int n_heads = q.sizes()[2]; 
	int n_ctx = q.sizes()[3]; 
	
	double scale = -1.0 / sqrt(width);
	
	auto options = torch::TensorOptions()
		.dtype(q.dtype())
		.device(q.device())
		.requires_grad(q.requires_grad());
	
	auto d_q = torch::zeros({bs, n_ctx, n_heads, width}, options);
	auto d_k = torch::zeros({bs, n_ctx, n_heads, width}, options);
	
	const dim3 numBlocks(n_heads*n_ctx, width, bs); // x, y, z
	const dim3 threadsPerBlock(32, 1, 1);
	
	AT_DISPATCH_FLOATING_TYPES(q.scalar_type(), "l1attn_cuda_backward_kernel", ([&] {
		l1attn_cuda_backward_kernel<scalar_t><<<numBlocks, threadsPerBlock>>>(
			d_attnq.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			d_attnk.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			q.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			k.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			d_q.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			d_k.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			scale, bs, n_ctx, n_heads, width);
	}));
	
	return {d_q, d_k};
}

std::vector<torch::Tensor> l1attn_cuda_backward16(
		torch::Tensor d_attn,
		torch::Tensor q,
		torch::Tensor k) 
{
	int bs = q.sizes()[0]; 
	int n_heads = q.sizes()[1]; 
	int n_ctx = q.sizes()[2]; 
	int width = q.sizes()[3];
	
	double scale = -1.0 / sqrt(width);
	int zwidth = n_ctx / 16; 
	
	auto options = torch::TensorOptions()
		.dtype(q.dtype())
		.device(q.device())
		.requires_grad(q.requires_grad());
	
	auto d_q = torch::zeros({bs, n_heads, n_ctx, width}, options);
	auto d_k = torch::zeros({bs, n_heads, n_ctx, width}, options);
	
	// const dim3 dimBlocks(32, 8); // x, y, z
	const dim3 numBlocks(zwidth, zwidth, n_heads*bs); // x, y, z
	const dim3 threadsPerBlock(16, 16, 1); 
	
	AT_DISPATCH_FLOATING_TYPES(q.scalar_type(), "l1attn_cuda_backward_kernel16", ([&] {
		l1attn_cuda_backward_kernel16<scalar_t><<<numBlocks, threadsPerBlock>>>(
			d_attn.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			q.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			k.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			d_q.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			d_k.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			scale, bs, n_ctx, n_heads, width);
	}));
	
	// bhtw -> bthw -- really need to change everything in the lib! 
	d_q = d_q.transpose_(1,2).contiguous();
	d_k = d_k.transpose_(1,2).contiguous(); 
	
	return {d_q, d_k}; // reduce along the zsize dim
}
