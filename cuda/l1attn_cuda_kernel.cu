#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/native/cuda/KernelUtils.cuh>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

template <typename scalar_t>
__device__  __forceinline__ scalar_t sign(scalar_t x)
{ 
	scalar_t t = x < 0 ? -1 : 0;
	return x > 0 ? 1 : t;
}

template <typename scalar_t>
__device__  __forceinline__ void fastAtomicAdd2(
	torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> out, 
	int i0, int i1, int i2, int i3, scalar_t v)
{
	// convenience wrapper function around
	// fastAtomicAdd for 4-D tensors. 
	int index = i0*out.stride(0) + i1*out.stride(1) + i2*out.stride(2) + i3*out.stride(3);
	at::native::fastAtomicAdd(out.data(), index, 1, v, true); 
}

template <typename scalar_t>
__global__ void l1attn_cuda_forward_kernel(
		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		q,
		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		k,
		torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		attn,
		const scalar_t scale, 
		const int bs, const int n_ctx, const int n_heads, const int width)
{
	__shared__ scalar_t acc[32];
	
	int tix = threadIdx.x; // [0 .. 31]. 
	// tix operates within across the width dimension (reduction dim) 
	int h = blockIdx.x % n_heads; 
	int t = blockIdx.x / n_heads; 
	int s = blockIdx.y; 
	int b = blockIdx.z; 
	
	int width32 = (width + 31) / 32; 
	scalar_t f = 0.0; 
	for(int w = 0; w < width32; w++) { 
		int o = w*32+tix; 
		if(o < width)
			f += abs(q[b][t][h][o] - k[b][s][h][o]); 
	}
	acc[tix] = f * scale; 
	if(tix < 16) { 
		acc[tix] += acc[tix + 16];
		__syncthreads(); // why is this needed ??? 
		acc[tix] += acc[tix + 8 ];
		__syncthreads(); // threads in a warp should be synchronous.
		acc[tix] += acc[tix + 4 ];
		__syncthreads(); // experiment: it's totally needed! 
		acc[tix] += acc[tix + 2 ];
		__syncthreads();
		acc[tix] += acc[tix + 1 ];
		__syncthreads();
		if(tix == 0){
			attn[b][s][t][h] = acc[tix]; 
		}
	}
}

#define	BLKSIZ 16
template <typename scalar_t>
__global__ void l1attn_cuda_forward_kernel32(
		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		q,
		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		k,
		torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		attn,
		const scalar_t scale, 
		const int bs, const int n_ctx, const int n_heads, const int width)
{
	/* q and k must be bhtw and bhsw respectively
	 * this function operates on q and k tensors, in blocks of 16 x 16
	 * q and k must be width a multiple of 32 with a loop:
	 * Larger would require more per-warp memory or use of registers: 
	 * 2 x 16 x 32 x 4 bytes = 4096 kB per block, 
	 * so each SM can have 12 blocks = good
	 */
	
	int w = threadIdx.x; // t thread [0 .. 15]. 
	int u = threadIdx.y; // t for q, s for k,  [0 .. 15]. 
	int tb = blockIdx.x; // t block
	int sb = blockIdx.y; // s block
	int h = blockIdx.z % n_heads; // head
	int b = blockIdx.z / n_heads; // block
	
	// each block computes a BLKSIZ x BLKSIZ block of the attention matrix
	// a block is 256 threads
	// so, each thread loads one value from each q,k
	__shared__ scalar_t qc[BLKSIZ][32]; // q cache 
	__shared__ scalar_t kc[BLKSIZ][32]; // k cache
	
	//reshape to 8 warps, 32 threads - better mem throughput
	int tid = u*BLKSIZ + w; 
	int cw = tid % 32; // cache w (thread)
	int cu = tid / 32; // cache u (warp)
	
	scalar_t f = 0.0; 
	int t,s; 
	for(int wo = 0; wo < width; wo += 32){
		t = tb * BLKSIZ + cu; 
		s = sb * BLKSIZ + cu; 
		int cwo = cw + wo; 
		qc[cu  ][cw] = q[b][h][t][cwo]; // each thread reads/writes 4 fp32
		qc[cu+8][cw] = q[b][h][t+8][cwo]; // some bubbles but eh
		kc[cu  ][cw] = k[b][h][s][cwo];
		kc[cu+8][cw] = k[b][h][s+8][cwo];
		
		__syncthreads();
		
		// simple approach: each thread computes one attention value
		// redefine t and s
		t = u; // so q is shared between threads in the same warp
		s = w; 
		for(int o=0; o < 32; o++){
			f += abs(qc[t][o] - kc[s][o]); // ultimately want these to be registers
		}
	}
	// back to global indexing
	t = tb * BLKSIZ + u; 
	s = sb * BLKSIZ + w; 
	attn[b][s][t][h] = f * scale; // this is unaligned. ought to fix.
}

#define	BLKSIZ 16
template <typename scalar_t>
__global__ void l1attn_cuda_forward_kernel64(
		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		q,
		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		k,
		torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		attn,
		const scalar_t scale, 
		const int bs, const int n_ctx, const int n_heads, const int width)
{
	/* q and k must be bhtw and bhsw respectively
	 * this function operates on q and k tensors, in blocks of 16 x 16
	 * q and k must be width a multiple of 32 with a loop:
	 * Larger would require more per-warp memory or use of registers: 
	 * 2 x 16 x 64 x 4 bytes = 8192 kB per block, 
	 * so each SM can have ~8 blocks = good
	 */
	
	int w = threadIdx.x; // t thread [0 .. 15]. 
	int u = threadIdx.y; // t for q, s for k,  [0 .. 15]. 
	int tb = blockIdx.x; // t block
	int sb = blockIdx.y; // s block
	int h = blockIdx.z % n_heads; // head
	int b = blockIdx.z / n_heads; // block
	
	// each block computes a BLKSIZ x BLKSIZ block of the attention matrix
	// a block is 256 threads
	// so, each thread loads one value from each q,k
	__shared__ scalar_t qc[BLKSIZ][64]; // q cache 
	__shared__ scalar_t kc[BLKSIZ][64]; // k cache
	
	//reshape to 8 warps, 32 threads - better mem throughput
	int tid = u*BLKSIZ + w; 
	int cw = tid % 32; // cache w (thread)
	int cu = tid / 32; // cache u (warp)
	
	scalar_t f = 0.0; 
	int t,s; 
	/* NOTE: width == 64 here  
	 * thereby avoiding branching or loops */
	t = tb * BLKSIZ + cu; 
	s = sb * BLKSIZ + cu; 
	qc[cu  ][cw] = q[b][h][t][cw]; // each thread reads/writes 4 fp32
	qc[cu  ][cw+32] = q[b][h][t][cw+32];
	qc[cu+8][cw] = q[b][h][t+8][cw]; // some bubbles but eh
	qc[cu+8][cw+32] = q[b][h][t+8][cw+32];
	kc[cu  ][cw] = k[b][h][s][cw];
	kc[cu  ][cw+32] = k[b][h][s][cw+32];
	kc[cu+8][cw] = k[b][h][s+8][cw];
	kc[cu+8][cw+32] = k[b][h][s+8][cw+32];
	
	__syncthreads();
	
	// simple approach: each thread computes one attention value
	// redefine t and s
	t = u; // so q is shared between threads in the same warp
	s = w; 
	for(int o=0; o < 64; o++){
		f += abs(qc[t][o] - kc[s][o]); // ultimately want these to be registers
	}
	// back to global indexing
	t = tb * BLKSIZ + u; 
	s = sb * BLKSIZ + w; 
	attn[b][s][t][h] = f * scale; // this is unaligned. ought to fix.
}


template <typename scalar_t>
__global__ void l1attn_cuda_backward_kernel(
		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		d_attnq,
		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		d_attnk,
		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		q,
		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		k,
		torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		d_q,
		torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		d_k,
		const scalar_t scale, 
		const int bs, const int n_ctx, const int n_heads, const int width ) 
{
	__shared__ scalar_t acc_dq[32];
	__shared__ scalar_t acc_dk[32];
	
	int tix = threadIdx.x; // [0 .. 31].
	int h = blockIdx.x % n_heads; 
	int r = blockIdx.x / n_heads; // r is t for q, s for k.
	int w = blockIdx.y; 
	int b = blockIdx.z; 
		
	int ctx32 = (n_ctx + 31) / 32; 
	scalar_t dq = 0.0; 
	scalar_t dk = 0.0; 
	
	scalar_t qq = q[b][w][h][r]; 
	for(int o = 0; o < ctx32; o++) { 
		int s = o*32+tix; 
		if(s < n_ctx){ 
			// all this would work better if n_ctx were a multiple of 32. 
			scalar_t ws = qq - k[b][w][h][s];
			ws = sign(ws) * scale; 
			scalar_t d_a = d_attnq[b][r][h][s]; 
			dq += ws * d_a; 
		}
	}
	
	scalar_t kk = k[b][w][h][r]; 
	for(int o = 0; o < ctx32; o++) { 
		int t = o*32+tix; 
		if(t < n_ctx){
			scalar_t ws = q[b][w][h][t] - kk;
			ws = sign(ws) * scale; 
			scalar_t d_a = d_attnk[b][r][h][t]; 
			dk -= ws * d_a; 
		}
	}
	
	acc_dq[tix] = dq;
	acc_dk[tix] = dk;
	if(tix < 16) { 
		acc_dq[tix] += acc_dq[tix + 16];
		acc_dk[tix] += acc_dk[tix + 16];
		__syncthreads(); 
		acc_dq[tix] += acc_dq[tix + 8 ];
		acc_dk[tix] += acc_dk[tix + 8 ];
		__syncthreads(); 
		acc_dq[tix] += acc_dq[tix + 4 ];
		acc_dk[tix] += acc_dk[tix + 4 ];
		__syncthreads();
		acc_dq[tix] += acc_dq[tix + 2 ];
		acc_dk[tix] += acc_dk[tix + 2 ];
		__syncthreads();
		acc_dq[tix] += acc_dq[tix + 1 ];
		acc_dk[tix] += acc_dk[tix + 1 ];
		__syncthreads();
		if(tix == 0){
			d_q[b][r][h][w] = acc_dq[tix];
			d_k[b][r][h][w] = acc_dk[tix]; 
		}
	}
}

template <typename scalar_t>
__global__ void l1attn_cuda_backward_kernel32(
		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		d_attn,
		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		q,
		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		k,
		torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		d_q,
		torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		d_k,
		const scalar_t scale, 
		const int bs, const int n_ctx, const int n_heads, const int width) 
{
	// q and k must be bhtw and bhsw respectively
	// d_attn must be bhts (usually bsth)
	// output is bhtzw / bhszw, where z is an extra reduction dim over 16x16 
	
	int v = threadIdx.x; // thread [0 .. 15]. 
	int r = threadIdx.y; // t for q, s for k,  [0 .. 15]. 
	int sb = blockIdx.x; // s block
	int tb = blockIdx.y; // t block
	int h = blockIdx.z % n_heads; // head
	int b = blockIdx.z / n_heads; // block
	
	// each block computes a BLKSIZ x 32 block of d_q, d_k
	// a block is 256 threads
	// so, each thread loads four values from each q,k
	// and one from d_attn
	__shared__ scalar_t dac[BLKSIZ][BLKSIZ]; // d_attn cache 
	__shared__ scalar_t qc[BLKSIZ][32]; // q cache 
	__shared__ scalar_t kc[BLKSIZ][32]; // k cache
	
	// this will be partly uncoalesced w/ BLKSIZ=16
	int s = sb * BLKSIZ + v; 
	int t = tb * BLKSIZ + r; 
	dac[r][v] = d_attn[b][h][t][s]; 
	
	int tid = r*BLKSIZ + v; 
	int cw = tid % 32; // cache w (thread)
	int cr = tid / 32; // cache r (warp)
	
	for(int wo = 0; wo < width; wo += 32){
		t = tb * BLKSIZ + cr; 
		s = sb * BLKSIZ + cr; 
		int cwo = cw + wo; 
		qc[cr  ][cw] = q[b][h][t][cwo]; // each thread reads one fp32
		qc[cr+8][cw] = q[b][h][t+8][cwo];
		kc[cr  ][cw] = k[b][h][s][cwo]; // full 32-wide load
		kc[cr+8][cw] = k[b][h][s+8][cwo];
		__syncthreads();
		
		scalar_t dq, dk, qq, kk;
		for(int p = 0; p < 32; p += 16){
			int cw = v + p; 
			dq = 0.0;
			t = r; 
			qq = qc[t][cw]; 
			for(s = 0; s < BLKSIZ; s++){
				scalar_t ws = qq - kc[s][cw];
				ws = sign(ws) * scale; 
				dq += ws * dac[t][s]; 
			}
			t = tb * BLKSIZ + r;
			//d_q[b][t][h][z][w] = dq; 
			fastAtomicAdd2( d_q, b,h,t,cw+wo, dq ); 
			// TODO: add another cache level for this.  
			// will be write cache in the same way above is read-cache. 
		
			dk = 0.0; 
			s = r; 
			kk = kc[s][cw]; 
			for(t = 0; t < BLKSIZ; t++){
				scalar_t ws = qc[t][cw] - kk;
				ws = sign(ws) * scale; 
				dk -= ws * dac[t][s]; 
			}
			s = sb * BLKSIZ + r; 
			//d_k[b][s][h][z][w] = dk; 
			fastAtomicAdd2( d_k, b,h,s,cw+wo, dk ); 
		}
	}
}

template <typename scalar_t>
__global__ void l1attn_cuda_backward_kernel64(
		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		d_attn,
		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		q,
		const torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		k,
		torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		d_q,
		torch::PackedTensorAccessor32<scalar_t,4,torch::RestrictPtrTraits> 
		d_k,
		const scalar_t scale, 
		const int bs, const int n_ctx, const int n_heads, const int width) 
{
	// q and k must be bhtw and bhsw respectively
	// d_attn must be bhts (usually bsth)
	// output is bhtzw / bhszw, where z is an extra reduction dim over 16x16 
	
	int v = threadIdx.x; // thread [0 .. 15]. 
	int r = threadIdx.y; // t for q, s for k,  [0 .. 15]. 
	int sb = blockIdx.x; // s block
	int tb = blockIdx.y; // t block
	int h = blockIdx.z % n_heads; // head
	int b = blockIdx.z / n_heads; // block
	
	// each block computes a BLKSIZ x 32 block of d_q, d_k
	// a block is 256 threads
	// so, each thread loads four values from each q,k
	// and one from d_attn
	__shared__ scalar_t dac[BLKSIZ][BLKSIZ]; // d_attn cache 
	__shared__ scalar_t qc[BLKSIZ][64]; // q cache 
	__shared__ scalar_t kc[BLKSIZ][64]; // k cache
	
	// this will be partly uncoalesced w/ BLKSIZ=16
	int s = sb * BLKSIZ + v; 
	int t = tb * BLKSIZ + r; 
	dac[r][v] = d_attn[b][h][t][s]; 
	
	int tid = r*BLKSIZ + v; 
	int cw = tid % 32; // cache w (thread)
	int cr = tid / 32; // cache r (warp)
	
	/* NOTE: width == 64 here  
	 * thereby avoiding branching or loops */
	t = tb * BLKSIZ + cr; 
	s = sb * BLKSIZ + cr;  
	qc[cr  ][cw] = q[b][h][t][cw]; // each thread reads one fp32
	qc[cr  ][cw+32] = q[b][h][t][cw+32];
	qc[cr+8][cw] = q[b][h][t+8][cw];
	qc[cr+8][cw+32] = q[b][h][t+8][cw+32];
	kc[cr  ][cw] = k[b][h][s][cw]; // full 32-wide load
	kc[cr  ][cw+32] = k[b][h][s][cw+32];
	kc[cr+8][cw] = k[b][h][s+8][cw];
	kc[cr+8][cw+32] = k[b][h][s+8][cw+32];
	__syncthreads();
	
	scalar_t dq, dk, qq, kk;
	for(int p = 0; p < 64; p += 16){
		cw = v + p; 
		dq = 0.0;
		t = r; 
		qq = qc[t][cw]; 
		for(s = 0; s < BLKSIZ; s++){
			scalar_t ws = qq - kc[s][cw];
			ws = sign(ws) * scale; 
			dq += ws * dac[t][s]; 
		}
		t = tb * BLKSIZ + r;
		//d_q[b][t][h][z][w] = dq; 
		fastAtomicAdd2( d_q, b,h,t,cw, dq ); 
		// TODO: add another cache level for this.  
		// will be write cache in the same way above is read-cache. 
	
		dk = 0.0; 
		s = r; 
		kk = kc[s][cw]; 
		for(t = 0; t < BLKSIZ; t++){
			scalar_t ws = qc[t][cw] - kk;
			ws = sign(ws) * scale; 
			dk -= ws * dac[t][s]; 
		}
		s = sb * BLKSIZ + r; 
		//d_k[b][s][h][z][w] = dk; 
		fastAtomicAdd2( d_k, b,h,s,cw, dk ); 
	}
}

std::vector<torch::Tensor> l1attn_cuda_forward(
		torch::Tensor q,
		torch::Tensor k) {
  
	int bs = q.sizes()[0]; 
	int n_ctx = q.sizes()[1]; 
	int n_heads = q.sizes()[2]; 
	int width = q.sizes()[3];
	
	auto options = torch::TensorOptions()
		.dtype(q.dtype())
		.device(q.device())
		.requires_grad(q.requires_grad()); //better way to do this? 
	
	auto attn = torch::zeros({bs, n_ctx, n_ctx, n_heads}, options); 
	
	const dim3 numBlocks(n_heads*n_ctx, n_ctx, bs); // x, y, z
	const dim3 threadsPerBlock(32, 1, 1);
	
	double scale = -1.0 / sqrt(width); 
		
	AT_DISPATCH_FLOATING_TYPES(q.scalar_type(), "l1attn_cuda_forward_kernel", ([&] {
		l1attn_cuda_forward_kernel<scalar_t><<<numBlocks, threadsPerBlock>>>(
			q.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			k.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			attn.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			scale, bs, n_ctx, n_heads, width);
	}));
	
	return {attn};
}

std::vector<torch::Tensor> l1attn_cuda_forward32(
		torch::Tensor q,
		torch::Tensor k) {
  
	int bs = q.sizes()[0]; 
	int n_heads = q.sizes()[1];
	int n_ctx = q.sizes()[2]; 
	int width = q.sizes()[3];
	
	auto options = torch::TensorOptions()
		.dtype(q.dtype())
		.device(q.device())
		.requires_grad(q.requires_grad()); //better way to do this? 
	
	auto attn = torch::zeros({bs, n_ctx, n_ctx, n_heads}, options); 
	
	const dim3 numBlocks(n_ctx/BLKSIZ, n_ctx/BLKSIZ, bs*n_heads); // x, y, z
	const dim3 threadsPerBlock(BLKSIZ, BLKSIZ, 1);
	
	double scale = -1.0 / sqrt(width); 
		
	AT_DISPATCH_FLOATING_TYPES(q.scalar_type(), "l1attn_cuda_forward_kernel32", ([&] {
		l1attn_cuda_forward_kernel32<scalar_t><<<numBlocks, threadsPerBlock>>>(
			q.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			k.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			attn.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			scale, bs, n_ctx, n_heads, width);
	}));
	
	return {attn};
}

std::vector<torch::Tensor> l1attn_cuda_forward64(
		torch::Tensor q,
		torch::Tensor k) {
  
	int bs = q.sizes()[0]; 
	int n_heads = q.sizes()[1];
	int n_ctx = q.sizes()[2]; 
	int width = q.sizes()[3];
	
	auto options = torch::TensorOptions()
		.dtype(q.dtype())
		.device(q.device())
		.requires_grad(q.requires_grad()); //better way to do this? 
	
	auto attn = torch::zeros({bs, n_ctx, n_ctx, n_heads}, options); 
	
	const dim3 numBlocks(n_ctx/BLKSIZ, n_ctx/BLKSIZ, bs*n_heads); // x, y, z
	const dim3 threadsPerBlock(BLKSIZ, BLKSIZ, 1);
	
	double scale = -1.0 / sqrt(width); 
		
	AT_DISPATCH_FLOATING_TYPES(q.scalar_type(), "l1attn_cuda_forward_kernel64", ([&] {
		l1attn_cuda_forward_kernel64<scalar_t><<<numBlocks, threadsPerBlock>>>(
			q.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			k.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			attn.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			scale, bs, n_ctx, n_heads, width);
	}));
	
	return {attn};
}

std::vector<torch::Tensor> l1attn_cuda_backward(
		torch::Tensor d_attnq,
		torch::Tensor d_attnk,
		torch::Tensor q,
		torch::Tensor k) 
{
	int bs = q.sizes()[0]; // permuted in python driver!!!
	int width = q.sizes()[1];
	int n_heads = q.sizes()[2]; 
	int n_ctx = q.sizes()[3]; 
	
	double scale = -1.0 / sqrt(width);
	
	auto options = torch::TensorOptions()
		.dtype(q.dtype())
		.device(q.device())
		.requires_grad(q.requires_grad());
	
	auto d_q = torch::zeros({bs, n_ctx, n_heads, width}, options);
	auto d_k = torch::zeros({bs, n_ctx, n_heads, width}, options);
	
	const dim3 numBlocks(n_heads*n_ctx, width, bs); // x, y, z
	const dim3 threadsPerBlock(32, 1, 1);
	
	AT_DISPATCH_FLOATING_TYPES(q.scalar_type(), "l1attn_cuda_backward_kernel", ([&] {
		l1attn_cuda_backward_kernel<scalar_t><<<numBlocks, threadsPerBlock>>>(
			d_attnq.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			d_attnk.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			q.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			k.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			d_q.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			d_k.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			scale, bs, n_ctx, n_heads, width);
	}));
	
	return {d_q, d_k};
}

std::vector<torch::Tensor> l1attn_cuda_backward32(
		torch::Tensor d_attn,
		torch::Tensor q,
		torch::Tensor k) 
{
	int bs = q.sizes()[0]; 
	int n_heads = q.sizes()[1]; 
	int n_ctx = q.sizes()[2]; 
	int width = q.sizes()[3];
	
	double scale = -1.0 / sqrt(width);
	int zwidth = n_ctx / 16; 
	
	auto options = torch::TensorOptions()
		.dtype(q.dtype())
		.device(q.device())
		.requires_grad(q.requires_grad());
	
	auto d_q = torch::zeros({bs, n_heads, n_ctx, width}, options);
	auto d_k = torch::zeros({bs, n_heads, n_ctx, width}, options);
	
	// const dim3 dimBlocks(32, 8); // x, y, z
	const dim3 numBlocks(zwidth, zwidth, n_heads*bs); // x, y, z
	const dim3 threadsPerBlock(16, 16, 1); 
	
	AT_DISPATCH_FLOATING_TYPES(q.scalar_type(), "l1attn_cuda_backward_kernel32", ([&] {
		l1attn_cuda_backward_kernel32<scalar_t><<<numBlocks, threadsPerBlock>>>(
			d_attn.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			q.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			k.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			d_q.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			d_k.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			scale, bs, n_ctx, n_heads, width);
	}));
	
	// bhtw -> bthw -- really need to change everything in the lib! 
	d_q = d_q.transpose_(1,2).contiguous();
	d_k = d_k.transpose_(1,2).contiguous(); 
	
	return {d_q, d_k}; // reduce along the zsize dim
}

std::vector<torch::Tensor> l1attn_cuda_backward64(
		torch::Tensor d_attn,
		torch::Tensor q,
		torch::Tensor k) 
{
	int bs = q.sizes()[0]; 
	int n_heads = q.sizes()[1]; 
	int n_ctx = q.sizes()[2]; 
	int width = q.sizes()[3];
	
	double scale = -1.0 / sqrt(width);
	int zwidth = n_ctx / 16; 
	
	auto options = torch::TensorOptions()
		.dtype(q.dtype())
		.device(q.device())
		.requires_grad(q.requires_grad());
	
	auto d_q = torch::zeros({bs, n_heads, n_ctx, width}, options);
	auto d_k = torch::zeros({bs, n_heads, n_ctx, width}, options);
	
	// const dim3 dimBlocks(32, 8); // x, y, z
	const dim3 numBlocks(zwidth, zwidth, n_heads*bs); // x, y, z
	const dim3 threadsPerBlock(16, 16, 1); 
	
	AT_DISPATCH_FLOATING_TYPES(q.scalar_type(), "l1attn_cuda_backward_kernel64", ([&] {
		l1attn_cuda_backward_kernel64<scalar_t><<<numBlocks, threadsPerBlock>>>(
			d_attn.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			q.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			k.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			d_q.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			d_k.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
			scale, bs, n_ctx, n_heads, width);
	}));
	
	// bhtw -> bthw -- really need to change everything in the lib! 
	d_q = d_q.transpose_(1,2).contiguous();
	d_k = d_k.transpose_(1,2).contiguous(); 
	
	return {d_q, d_k}; // reduce along the zsize dim
}
